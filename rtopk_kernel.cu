#include "hip/hip_runtime.h"
#include "rtopk_kernel.cuh"

template <int WARPS_PER_BLOCK>
__global__ void rtopk_kernel(float *data, float *value, int *index, int N, int dim_origin, int k, int max_iter, float precision)
{
    extern __shared__ float cache[]; 
    const int wid = threadIdx.x / 32;
    const int laneid = threadIdx.x % 32;

    if (blockIdx.x * WARPS_PER_BLOCK + wid >= N){
        return;
    }

    const int dim_len = (dim_origin + 31) / 32;

    const int first_idx = blockIdx.x * WARPS_PER_BLOCK * dim_origin;
    #pragma unroll
    for(int ext = 0; ext < dim_len; ext++){
        int data_idx = wid * dim_origin + laneid + ext * 32;
        if (data_idx < dim_origin){
            cache[data_idx] = data[first_idx + data_idx];
        }
    }

    __syncwarp();

    float max_data = -99999, min_data = 99999;

    #pragma unroll
    for(int j = 0; j < dim_len; j++){
        int data_idx = wid * dim_origin + laneid  + j * 32;
        if (data_idx >= dim_origin) {
            break;
        }
        if(cache[data_idx] > max_data){
            max_data = cache[data_idx];
        }
        if(cache[data_idx] < min_data){
            min_data = cache[data_idx];
        }
    }

    #pragma unroll
    for (int offset = 32 / 2; offset > 0; offset /= 2) {
        max_data = max(max_data, __shfl_down_sync(0xFFFFFFFF, max_data, offset));
        min_data = min(min_data, __shfl_down_sync(0xFFFFFFFF, min_data, offset));
    }

    max_data = __shfl_sync(0xFFFFFFFF, max_data, 0);
    min_data = __shfl_sync(0xFFFFFFFF, min_data, 0);

    float mid_data = max_data;

    int count;

    for(int i = 0; ; i++){
        count = 0;
        #pragma unroll
        for(int j = 0; j < dim_len; j++){
            int data_idx = wid * dim_origin + laneid  + j * 32;
            if (data_idx < dim_origin) {
                count += cache[data_idx] >= mid_data;
            }
        }
        count += __shfl_down_sync(0xffffffff, count, 16);
        count += __shfl_down_sync(0xffffffff, count, 8);
        count += __shfl_down_sync(0xffffffff, count, 4);
        count += __shfl_down_sync(0xffffffff, count, 2);
        count += __shfl_down_sync(0xffffffff, count, 1);
        count = __shfl_sync(0xffffffff, count, 0);

        if(i >= max_iter || mid_data <= min_data + precision){
            break;
        }

        if(count < k){
            max_data = mid_data;
        }
        else if(count > k){
            min_data = mid_data;
        }
        else{
            break;
        }        
        mid_data = (min_data + max_data) / 2;        
    }

    int eq_n = k - count; 
    int total_cnt = 0, total_cnt_eq = 0, total_cnt_whole = 0;

    #pragma unroll
    for(int ext = 0; ext < dim_len; ext++){
        if(total_cnt_whole >= k){
            break;
        }
        int data_idx = wid * dim_origin + laneid + ext * 32;
        if (data_idx >= dim_origin) {
            break;
        }
        float val = cache[data_idx];
        bool choose = val >= mid_data;

        bool choose_eq = val >= min_data && val < mid_data;

        unsigned mask = __ballot_sync(0xffffffff, choose); 
        unsigned mask_eq = __ballot_sync(0xffffffff, choose_eq);

        int lane_cnt = __popc(mask & ((1 << (laneid + 1)) - 1));
        int lane_cnt_eq = __popc(mask_eq & ((1 << (laneid + 1)) - 1));

        if (total_cnt + lane_cnt > k) {
            choose = 0;
        }       
        if (total_cnt_eq + lane_cnt_eq > eq_n ){
            choose_eq = 0;
        }

        mask = __ballot_sync(0xffffffff, choose);
        mask_eq = __ballot_sync(0xffffffff, choose_eq);

        unsigned mask_whole = mask | mask_eq;

        lane_cnt = __popc(mask & ((1 << (laneid + 1)) - 1));
        lane_cnt_eq = __popc(mask_eq & ((1 << (laneid + 1)) - 1));
        int lane_cnt_whole = __popc(mask_whole & ((1 << (laneid + 1)) - 1));

        if(choose || choose_eq){
            value[blockIdx.x * WARPS_PER_BLOCK * k + wid * k + total_cnt_whole + lane_cnt_whole - 1] = val;
            index[blockIdx.x * WARPS_PER_BLOCK * k + wid * k + total_cnt_whole + lane_cnt_whole - 1] = laneid + ext * 32;
        }

        total_cnt += lane_cnt;
        total_cnt = __shfl_sync(0xffffffff, total_cnt, 31);

        total_cnt_eq += lane_cnt_eq;
        total_cnt_eq = __shfl_sync(0xffffffff, total_cnt_eq, 31);

        total_cnt_whole += lane_cnt_whole;
        total_cnt_whole = __shfl_sync(0xffffffff, total_cnt_whole, 31);

    }
}


template __global__ void rtopk_kernel<8>(float*, float*, int*, int, int, int, int, float);
template __global__ void rtopk_kernel<4>(float*, float*, int*, int, int, int, int, float);
template __global__ void rtopk_kernel<2>(float*, float*, int*, int, int, int, int, float);
template __global__ void rtopk_kernel<1>(float*, float*, int*, int, int, int, int, float);